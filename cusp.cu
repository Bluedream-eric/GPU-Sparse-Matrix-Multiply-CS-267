#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <cusp/csr_matrix.h>
#include <cusp/array2d.h>
#include <cusp/array1d.h>
#include <cusp/multiply.h>
#include <cusp/gallery/random.h>
#include <cusp/gallery/poisson.h>
#include <cusp/functional.h>
#include <cusp/print.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <random>
#include <map>

#define P_SIZE 14

using namespace std;

thrust::identity<int> zero;
thrust::multiplies<int> combine;
thrust::plus<int> reduce;
ofstream outputFile;

const double p[] = {0.01, 0.02, 0.03, 0.04, 0.05, 0.1, 0.15, 0.2, 0.25, 0.3, 0.35, 0.4, 0.45, 0.5};

int csrTimesCsr() {
        outputFile.open("cusp_csvs/cusp_csr_csr.csv");
        for (int i = 4; i < 15; i++) {
                int N = pow(2, i);
                for (int j = 0; j < P_SIZE; j++) {
                        cusp::csr_matrix<int, int, cusp::device_memory> X_GPU;
                        cusp::csr_matrix<int, int, cusp::device_memory> Y_GPU;
                        int num_samples = ((int) (p[j] * N * N));
                        cout << "N is: " << N << " density is " << p[j];
                        cout << "\nNum Samples: " << num_samples << "\n";
                        cusp::gallery::random(X_GPU, N, N, num_samples);
                        cusp::gallery::random(Y_GPU, N, N, num_samples);
                        cusp::csr_matrix<int, int, cusp::device_memory> Z;
                        clock_t start = clock();
                        cusp::generalized_spgemm(X_GPU, Y_GPU, Z, zero, combine, reduce);
                        clock_t diff = clock() - start;
                        float sec = ((float) diff) / CLOCKS_PER_SEC;
                        outputFile << N << "," << p[j] << "," << std::setprecision(15) << sec << "\n";
                }
        }
        outputFile.close();
        return 1;
}

int csrTimesDenseVector() {
         ofstream outputFile;
         outputFile.open("cusp_csvs/cusp_csr_denseVec.csv");
         for(int i = 4; i < 15; i++) {
                 int N = pow(2, i);
                 for(int j = 0; j < P_SIZE; j++) {
                     cusp::csr_matrix<int, int, cusp::device_memory> X_GPU;
                     cusp::csr_matrix<int, int, cusp::device_memory> Y_GPU;
                     int num_samples = ((int) (p[j] * N * N));
                     int randomNum = rand() % N;
                     cout << "N is: " << N << " density is " << p[j];
                     cout << "\nNum Samples: " << num_samples << "\n";
                     cusp::gallery::random(X_GPU, N, N, num_samples);
                     cusp::gallery::random(Y_GPU, N, 1, randomNum);
                     cusp::csr_matrix<int, int, cusp::device_memory> Z;
                     clock_t start = clock();
                     cusp::generalized_spgemm(X_GPU, Y_GPU, Z, zero, combine, reduce);
                     clock_t diff = clock() - start;
                     float sec = ((float) diff) / CLOCKS_PER_SEC;
                     outputFile << N << "," << p[j] << "," << std::setprecision(15) << sec << "\n";
                 }
         }
         outputFile.close();
         return 1;
}

int csrTimesDenseMatrix() {
    typedef cusp::csr_matrix<int, float, cusp::device_memory>             Matrix;
    typedef cusp::array1d<float, cusp::device_memory>                     Array1d;
    typedef cusp::array2d<float, cusp::device_memory, cusp::column_major> Array2d;
    typedef Array2d::column_view column_view;

    ofstream outputFile;
    outputFile.open("cusp_csvs/cusp_csr_denseMat.csv");

    for (int i = 4; i < 15; i++) {
        int N = pow(2, i);
        for(int j = 0; j < P_SIZE; j++) {
            Matrix A;
            int num_samples = ((int) (p[j] * N * N));
            int randomNum = rand() % N;
            cusp::gallery::random(A, N, N, num_samples);

            Array2d X(N, N);
            cusp::gallery::random(X, N, N, randomNum);
            Array1d y(N);

            column_view x = X.column(0);
            cout << "N is: " << N << " density is " << p[j];
            cout << "\nNum Samples: " << num_samples << "\n";
            clock_t start = clock();
            cusp::multiply(A, x, y);
            clock_t diff = clock() - start;
            float sec = ((float) diff) / CLOCKS_PER_SEC;
            outputFile << N << "," << p[j] << "," << std::setprecision(15) << sec << "\n";
        }
    }
    return 1;
}




int main() {
        csrTimesCsr();
        csrTimesDenseVector();
        csrTimesDenseMatrix();
        system("sudo shutdown -P now");
}
