#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <cusp/csr_matrix.h>
#include <cusp/array2d.h>
#include <cusp/array1d.h>
#include <cusp/multiply.h>
#include <cusp/gallery/random.h>
#include <cusp/print.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <random>
#include <map>

#define P_SIZE 14

using namespace std;

const double p[] = {0.01, 0.02, 0.03, 0.04, 0.05, 0.1, 0.15, 0.2, 0.25, 0.3, 0.35, 0.4, 0.45, 0.5};

int csrTimesCsr() {
        ofstream outputFile;
        outputFile.open("cusp_csvs/cusp_csr_csr.csv");
        for (int i = 4; i < 15; i++) {
                int N = pow(2, i);
                for (int j = 0; j < P_SIZE; j++) {
                        cusp::csr_matrix<int, int, cusp::device_memory> X_GPU;
                        cusp::csr_matrix<int, int, cusp::device_memory> Y_GPU;
                        int num_samples = ((int) (p[j] * N * N));
                        cout << "N is: " << N << " density is " << p[j];
                        cout << "\nNum Samples: " << num_samples << "\n";
                        cusp::gallery::random(N, N, num_samples, X_GPU);
                        cusp::gallery::random(N, N, num_samples, Y_GPU);
                        cusp::csr_matrix<int, int, cusp::device_memory> Z;
                        clock_t start = clock();
                        cusp::multiply(X_GPU, Y_GPU, Z);
                        clock_t diff = clock() - start;
                        float sec = ((float) diff) / CLOCKS_PER_SEC;
                        outputFile << N << "," << p[j] << "," << std::setprecision(15) << sec << "\n";
                }
        }
        outputFile.close();
        return 1;
}

int sparseDTimesSparseD() {
        ofstream outputFile;
        outputFile.open("cusp_csvs/cusp_spD_spD.csv");
        for (int i = 4; i < 15; i++) {
                int N = pow(2, i);
                for (int j = 0; j < P_SIZE; j++) {
                        cusp::csr_matrix<int, int, cusp::device_memory> X_GPU;
                        cusp::csr_matrix<int, int, cusp::device_memory> Y_GPU;
                        cusp::array2d<int, cusp::device_memory> X_D(N,N);
                        cusp::array2d<int, cusp::device_memory> Y_D(N,N);
                        int num_samples = ((int) (p[j] * N * N));
                        cout << "N is: " << N << " density is " << p[j];
                        cout << "\nNum Samples: " << num_samples << "\n";
                        cusp::gallery::random(N, N, num_samples, X_GPU);
                        cusp::gallery::random(N, N, num_samples, Y_GPU);
                        cusp::convert(X_GPU, X_D);
                        cusp::convert(Y_GPU, Y_D);
                        cusp::array2d<int, cusp::device_memory> Z(N,N);
                        clock_t start = clock();
                        cusp::multiply(X_D, Y_D, Z);
                        clock_t diff = clock() - start;
                        float sec = ((float) diff) / CLOCKS_PER_SEC;
                        outputFile << N << "," << p[j] << "," << std::setprecision(15) << sec << "\n";
                }
        }
        outputFile.close();
        return 1;
}

int csrTimesDense() {
        ofstream outputFile;
        outputFile.open("cusp_csvs/cusp_csr_spD.csv");
        for (int i = 4; i < 15; i++) {
                int N = pow(2, i);
                for (int j = 0; j < P_SIZE; j++) {
                        cusp::csr_matrix<int, int, cusp::device_memory> X_GPU;
                        cusp::csr_matrix<int, int, cusp::device_memory> Y_GPU;
                        cusp::array2d<int, cusp::device_memory> Y_D(N,N);
                        int num_samples = ((int) (p[j] * N * N));
                        cout << "N is: " << N << " density is " << p[j];
                        cout << "\nNum Samples: " << num_samples << "\n";
                        cusp::gallery::random(N, N, num_samples, X_GPU);
                        cusp::gallery::random(N, N, num_samples, Y_GPU);
                        cusp::convert(Y_GPU, Y_D);
                        cusp::array2d<int, cusp::device_memory> Z(N,N);
                        clock_t start = clock();
                        cusp::multiply(X_GPU, Y_D, Z);
                        clock_t diff = clock() - start;
                        float sec = ((float) diff) / CLOCKS_PER_SEC;
                        outputFile << N << "," << p[j] << "," << std::setprecision(15) << sec << "\n";
                }
        }
        outputFile.close();
        return 1;
}

int sparseDTimesDense() {
        ofstream outputFile;
        outputFile.open("cusp_csvs/cusp_spD_dense.csv");
        for (int i = 4; i < 15; i++) {
                int N = pow(2, i);
                for (int j = 0; j < P_SIZE; j++) {
                        cusp::csr_matrix<int, int, cusp::device_memory> X_GPU;
                        cusp::csr_matrix<int, int, cusp::device_memory> Y_GPU;
                        cusp::array2d<int, cusp::device_memory> X_D(N,N);
                        cusp::array2d<int, cusp::device_memory> Y_D(N,N);
                        int num_samples = ((int) (p[j] * N * N));
                        int randomNum = rand() % (N * N);
                        cout << "N is: " << N << " density is " << p[j];
                        cout << "\nNum Samples: " << num_samples << "\n";
                        cusp::gallery::random(N, N, num_samples, X_GPU);
                        cusp::gallery::random(N, N, randomNum, Y_GPU);
                        cusp::convert(X_GPU, X_D);
                        cusp::convert(Y_GPU, Y_D);
                        cusp::array2d<int, cusp::device_memory> Z(N,N);
                        clock_t start = clock();
                        cusp::multiply(X_D, Y_D, Z);
                        clock_t diff = clock() - start;
                        float sec = ((float) diff) / CLOCKS_PER_SEC;
                        outputFile << N << "," << p[j] << "," << std::setprecision(15) << sec << "\n";
                }
        }
        outputFile.close();
        return 1;
}

int csrTimesDenseVector() {
        ofstream outputFile;
        outputFile.open("cusp_csvs/cusp_csr_denseVec.csv");
        for(int i = 4; i < 15; i++) {
                int N = pow(2, i);
                for(int j = 0; j < P_SIZE; j++) {
                        cusp::csr_matrix<int, int, cusp::device_memory> X_GPU;
                        cusp::array1d<int, cusp::device_memory> Y(N);
                        cusp::array1d<int, cusp::device_memory> Z(N);
                        int num_samples = ((int) (p[j] * N * N));
                        int randomNum = rand() % N;
                        cout << "N is: " << N << " density is " << p[j];
                        cout << "\nNum Samples: " << num_samples << "\n";
                        cusp::gallery::random(N, N, num_samples, X_GPU);
                        cusp::gallery::random(N, 1, randomNum, Y);
                        clock_t start = clock();
                        cusp::multiply(X_GPU, Y, Z);
                        clock_t diff = clock() - start;
                        float sec = ((float) diff) / CLOCKS_PER_SEC;
                        outputFile << N << "," << p[j] << "," << std::setprecision(15) << sec << "\n";
                }
        }
        outputFile.close();
        return 1;
}

int sparseDTimesDenseVector() {
        ofstream outputFile;
        outputFile.open("cusp_csvs/cusp_sparseD_denseVec.csv");
        for(int i = 4; i < 15; i++) {
                int N = pow(2, i);
                for(int j = 0; j < P_SIZE; j++) {
                        cusp::csr_matrix<int, int, cusp::device_memory> X_GPU;
                        cusp::array1d<int, cusp::device_memory> Y(N);
                        cusp::array1d<int, cusp::device_memory> Z(N);
                        cusp::array2d<int, cusp::device_memory> X_D(N,N);
                        int num_samples = ((int) (p[j] * N * N));
                        int randomNum = rand() % N;
                        cout << "N is: " << N << " density is " << p[j];
                        cout << "\nNum Samples: " << num_samples << "\n";
                        cusp::gallery::random(N, N, num_samples, X_GPU);
                        cusp::convert(X_GPU, X_D);
                        cusp::gallery::random(N, 1, randomNum, Y);
                        clock_t start = clock();
                        cusp::multiply(X_D, Y, Z);
                        clock_t diff = clock() - start;
                        float sec = ((float) diff) / CLOCKS_PER_SEC;
                        outputFile << N << "," << p[j] << "," << std::setprecision(15) << sec << "\n";
                }
        }
        outputFile.close();
        return 1;
}

int main() {
        csrTimesCsr();
        //system("shutdown -s");
}
ls
