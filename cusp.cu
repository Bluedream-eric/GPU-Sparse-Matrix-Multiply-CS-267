#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include <cusp/galley/random.h>
#include <cusp/print.h>
#include <iostream>
#include <random>

#define P_SIZE 14 

std::map<double, std::map<int, double>> my_map;

const double p[] = {0.01, 0.02, 0.03, 0.04, 0.05, 0.1, 0.15, 0.2, 0.25, 0.3, 0.35, 0.4, 0.45, 0.5};

void writeToCsv(std::string fname) {
	ofstream myFile;
	myFile.open(fname);
	myFile << ",16,32,64,128,256,512,1024,2048,4096,8192,16384\n"
	for(int i = 0; i < P_SIZE; i++) {
		std::map<int, double> inner_map = my_map[p[i]];
		myFile << p[i];
		myFile << ",";
		std::map<int, double>::iterator it;
		std::map<int, double>::iterator next;
		for(it = inner_map.begin(); it != inner_map.end(); it++) {
			myFile << it->second;
			next = it + 1;
			if(next != inner_map.end()) {
				myFile << ",";
			} else {
				myFile << "\n";
			}
		}
	}
	myFile.close();
}

int csrTimesCsr() {
	for (int i = 4; i < 15; i++) {
		int N = pow(2, i);
		for (int j = 0; j < P_SIZE; j++) {
			cusp::csr_matrix<int, int, cusp::device_memory> X_GPU;
			cusp::csr_matrix<int, int, cusp::device_memory> Y_GPU;
			int num_samples = ((int) p[j] * N * N);
			cusp::gallery::random(N, N, num_samples, X_GPU);
			cusp::gallery::random(N, N, num_samples, Y_GPU);
			cusp::array2d<int, cusp::device_memory> Z(N,N);
			clock_t start = clock();
			cusp::multiply(X_GPU, Y_GPU, Z);
			clock_t diff = clock() - start;
			double sec = ((double) diff) / CLOCKS_PER_SEC;
			if(my_map.count(p[j]) == 0) {
				my_map[p[j]] = {N: sec};
			} else {
				my_map[p[j]][N] = sec;
			}
		}
	}
	return 1;
}

int main() {
	csrTimesCsr();
	writeToCsv("cusp_sparseCSR_sparseCSR.csv");
}