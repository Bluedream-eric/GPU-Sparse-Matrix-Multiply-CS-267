#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include <cusp/gallery/random.h>
#include <cusp/print.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <random>
#include <map>

#define P_SIZE 14

using namespace std;

const double p[] = {0.01, 0.02, 0.03, 0.04, 0.05, 0.1, 0.15, 0.2, 0.25, 0.3, 0.35, 0.4, 0.45, 0.5};

int csrTimesCsr() {
        ofstream outputFile;
        outputFile.open("cusp_csr_csr.csv");
        for (int i = 4; i < 15; i++) {
                int N = pow(2, i);
                for (int j = 0; j < P_SIZE; j++) {
                        cusp::csr_matrix<int, int, cusp::device_memory> X_GPU;
                        cusp::csr_matrix<int, int, cusp::device_memory> Y_GPU;
                        int num_samples = ((int) (p[j] * N * N));
                        cout << "N is: " << N << " density is " << p[j];
                        cout << "\nNum Samples: " << num_samples << "\n";
                        cusp::gallery::random(N, N, num_samples, X_GPU);
                        cusp::gallery::random(N, N, num_samples, Y_GPU);
                        cusp::csr_matrix<int, int, cusp::device_memory> Z;
                        clock_t start = clock();
                        cusp::multiply(X_GPU, Y_GPU, Z);
                        clock_t diff = clock() - start;
                        float sec = ((float) diff) / CLOCKS_PER_SEC;
                        outputFile << N << "," << p[j] << "," << std::setprecision(15) << sec << "\n";
                }
        }
        return 1;
}

int main() {
        csrTimesCsr();
        //system("shutdown -s");
}
